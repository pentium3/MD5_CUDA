#include "hip/hip_runtime.h"
//findMessage.cpp

#include "stdafx.h"
#include "deviceMemoryDef.h"
#include "gpuMD5.h"

/**
��������
min��������С����
max��������󳤶�
searchScope�������ռ�
*/
pair<bool, string> findMessage(size_t min, size_t max, string searchScope) {
	bool isFound = false;
	size_t h_isFound = -1; size_t * d_isFound;    //���������ʶ 
	uchar* d_message; uchar h_message[16];	//���ģ����֧�ֳ���Ϊ16
	string message = "";
	
	//GoForce GT650M �Ƚ���������ã�1024*1024
	int nBlocks = 512;
	int nThreadsPerBlock = 512;
	size_t nTotalThreads = nBlocks * nThreadsPerBlock; // ���߳���
	size_t charsetLength = searchScope.length();  //�����ռ��ַ�������

	hipError_t error;
	error = hipMalloc((void**)&d_isFound, sizeof(size_t));
	if (error != hipSuccess){
		printCudaError(error,"���䣨���������ʶ���Դ����", __FILE__, __LINE__);
    }
	error = hipMemcpy(d_isFound, &h_isFound,  sizeof(size_t), hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printCudaError(error,"���������������ʶ�����Դ����", __FILE__, __LINE__);
    }
	error = hipMalloc((void**)&d_message, 16 * sizeof(uchar));
	if (error != hipSuccess){
		printCudaError(error,"����������������ģ��Դ����", __FILE__, __LINE__);
    }

	//����ÿ���̵߳�������ʼ��ַ
	float* h_startNumbers = new float[nTotalThreads];
	float* d_startNumbers;
	error = hipMalloc((void**)&d_startNumbers, nTotalThreads * sizeof(float));
	if (error != hipSuccess){
		printCudaError(error,"�����̵߳�������ʼ��ַ����", __FILE__, __LINE__);
    }

	for (size_t size = min; size <= max; ++size) {
		cout<<"��ǰ�������ȣ�"<<size<<endl;
		float maxValue = pow((float)charsetLength, (float)size);  //���ƥ����
		float nIterations = ceil(maxValue / (nBlocks * nThreadsPerBlock));//ÿ���̷߳����������,��ÿ���߳���Ҫ�����ĸ���
		for (size_t i = 0; i != nTotalThreads; ++i) {
		  h_startNumbers[i] = i * nIterations;
		}
		error = hipMemcpy(d_startNumbers, h_startNumbers, nTotalThreads * sizeof(float), hipMemcpyHostToDevice);
		if (error != hipSuccess){
			printCudaError(error,"���� �̵߳�������ʼ��ַ ���Դ����", __FILE__, __LINE__);
		}
		clock_t start = clock();
		//��ʼ����
		searchMD5<<< nBlocks, nThreadsPerBlock >>>(d_startNumbers, 
			nIterations, charsetLength, size, d_isFound, d_message);
    
		hipDeviceSynchronize();

		cout<<"��ʱ��"<<(clock()-start)/CLK_TCK<<endl;

		printf("%s\n", hipGetErrorString(hipGetLastError()));
		hipMemcpy(&h_isFound, d_isFound, sizeof(int), hipMemcpyDeviceToHost);
		printf("####################### h_isFound = %d\n", h_isFound);

		if (h_isFound != -1) {
		  printf("h_isFound=%d\n", h_isFound);
		  hipMemcpy(h_message, d_message, 16 * sizeof(uchar), hipMemcpyDeviceToHost);
        
		  for (size_t i = 0; i != size; ++i){
			message.push_back(h_message[i]);
		  }
		  isFound = true;
		  cout << message << endl;
		  break;
		}
	}


	//�ͷ��ڴ���Դ�
	hipFree(d_targetDigest);
	hipFree(d_powerSymbols);
	hipFree(d_powerValues);
	hipFree(d_isFound);
	hipFree(d_message);
	hipFree(d_startNumbers);

	delete(h_startNumbers);
	cout<<"�ͷ��ڴ����..."<<endl;
	return make_pair(isFound, message);
}